
#include <hip/hip_runtime.h>
#include "stdio.h"

#define CUDA_CHECK_ERROR(err)           \
if ((err) != hipSuccess) {          \
    printf("Cuda error: %s\n", hipGetErrorString(err));    \
    printf("Error in file: %s, line: %i\n", __FILE__, __LINE__);  \
}

__global__ void add_kernel(int* a, int* b, int* c){
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    c[i] = a[i] + b[i];
}

//void add(int* a, int* b, int* c, int n){
//    for(int i = 0; i < n; i++){
//        c[i] = a[i] + b[i];
//    }
//}

int main(){

    int n = 1 << 30;
    //host mem

    auto a = new int [n];
    auto b = new int [n];
    auto c = new int [n];

    for(int i = 0; i < n; i++){
        a[i] = i;
        b[i] = i;
        c[i] = 0;
    }
    //dev mem

    int *a_dev, *b_dev, *c_dev;

    CUDA_CHECK_ERROR(hipMalloc(&a_dev, n * sizeof(int)))
    CUDA_CHECK_ERROR(hipMalloc(&b_dev, n * sizeof(int)))
    CUDA_CHECK_ERROR(hipMalloc(&c_dev, n * sizeof(int)))

    //host to dev memcpy

    //cudaMemCpy(dst, src, count, type)
    CUDA_CHECK_ERROR(hipMemcpy(a_dev, a, n * sizeof(int), hipMemcpyHostToDevice))
    CUDA_CHECK_ERROR(hipMemcpy(b_dev, b, n * sizeof(int), hipMemcpyHostToDevice))
    CUDA_CHECK_ERROR(hipMemcpy(c_dev, c, n * sizeof(int), hipMemcpyHostToDevice))

    //kernel
    int threads = 1024;
    int block = (n + threads - 1) / threads;
    add_kernel <<<block, threads>>> (a_dev, b_dev, c_dev);


    //dev to host memcpy
    CUDA_CHECK_ERROR(hipMemcpy(c, c_dev, n * sizeof(int), hipMemcpyDeviceToHost))

    //print ans
    for(int i = 1 << 10; i < ((1 << 10) + 100); i++){
        printf("c[%i] = %i\n", i, c[i]);
    }
}